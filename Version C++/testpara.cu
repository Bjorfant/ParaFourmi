#include "hip/hip_runtime.h"
#include <thrust/for_each.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <iostream>
#include <vector>
#include <cstdlib>

#define taille 3

using namespace std;


 typedef enum {
	VIDE=0,
	ACCESSIBLE=1,
	ACCESSIBLE_CONFLIT=2,
	GRAIN_CONFLIT=3,
	GRAIN=4,
	FOURMI=5,
	TRANSIT=6
} State;


//retourne vrai si la case d'indice "index" est sur le bord gauche de la matrice

int isOnLeftBorder(int index) {
	return index%taille == 0;
}

//retourne vrai si la case d'indice "index" est sur le bord droit de la matrice 

int isOnRightBorder(int index) {
	return index%taille == taille - 1;
}

//retourne vrai si la case d'indice "index" est sur le bord sup�rieur de la matrice

int isOnTopBorder(int index) {
	return index%(taille*taille) - taille < 0;
}

//retourne vrai si la case d'indice "index" est sur le bord inf�rieur de la matrice

int isOnBottomBorder(int index) {
	return index%(taille*taille) + taille >= taille*taille;
}

//retourne vrai si la case d'indice "index" est sur le bord avant de la matrice 

int isOnFrontBorder(int index) {
	return index + (taille*taille) >= taille*taille*taille;
}

//retourne vrai si la case d'indice "index" est sur le bord arri�re de la matrice

int isOnBackBorder(int index) {
	return index < taille*taille;
}


struct moveIndex {

	const int delta, maxIndex;

	moveIndex(int _delta, int _maxIndex) : delta(_delta), maxIndex(_maxIndex) {}

	__host__ __device__
	int operator()(int index){
		return (index + delta)%maxIndex;
	}
};

struct isAccessible {

	template <typename Tuple>
	__host__ __device__
	void operator() (Tuple t) {
	
		int index = thrust::get<0>(t);
		int blocAtLeft = thrust::get<1>(t);
		int blocAtRight = thrust::get<2>(t);
		int blocAtTop = thrust::get<3>(t);
		int blocAtBottom = thrust::get<4>(t);
		int blocAtFront = thrust::get<5>(t);
		int blocAtBack = thrust::get<6>(t);
		
		// Calcul du nombre de voisins
		if (!isOnLeftBorder(index)) {
			if (blocAtLeft == GRAIN)
				thrust::get<7>(t) = true;
		}
		if (!isOnRightBorder(index)) {
			if (blocAtRight == GRAIN)
				thrust::get<7>(t) = true;
		}
		if (!isOnTopBorder(index)) {
			if (blocAtTop == GRAIN)
				thrust::get<7>(t) = true;
		}
		if (!isOnBottomBorder(index)) {
			if (blocAtBottom == GRAIN)
				thrust::get<7>(t) = true;
		}
		if (!isOnFrontBorder(index)) {
			if (blocAtFront == GRAIN)
				thrust::get<7>(t) = true;
		}
		if (!isOnBackBorder(index)) {
			if (blocAtBack == GRAIN)
				thrust::get<7>(t) = true;
		}
		thrust::get<7>(t) = false;
	}
};


int deplacement_alea(vector <int> voisins) {
	if (voisins.size() >= 1)
		return voisins[0];
	else
		return -1;
}

//r�cup�re les voisins d'une case de la matrice
//possibilit� de filtrer les voisins par une liste d'�tat que l'on cherche 
//si la liste est vide on renvoit tous les voisins
/*
vector <int> listeVoisins(int index, thrust::device_vector <int> filtre, thrust::host_vector<int> &matfourmi) {
	vector <int> voisins;
	bool all = filtre.empty(); //verifie s'il y a une condition
	if (!isOnRightBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index+1]) != filtre.end())
			voisins.push_back(index+1);
	if (!isOnLeftBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index-1]) != filtre.end())
			voisins.push_back(index-1);
	if (!isOnTopBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index-taille]) != filtre.end())
			voisins.push_back(index-taille);
	if (!isOnBottomBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index+taille]) != filtre.end())
			voisins.push_back(index+taille);
	if (!isOnFrontBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index+taille*taille]) != filtre.end())
			voisins.push_back(index+taille*taille);
	if (!isOnBackBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index-taille*taille]) != filtre.end())
			voisins.push_back(index-taille*taille);
	return voisins;
}

vector <int> listeVoisinsAccessibles(int index, thrust::host_vector<int> &matfourmi) {
	vector <int> v;
	v.push_back(ACCESSIBLE);
	return listeVoisins(index, v, matfourmi);
}*/


struct listeNbVoisinsActifs {

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t){

		int filtre = FOURMI;
		
		int index = thrust::get<0>(t);
		int blocAtLeft = thrust::get<1>(t);
		int blocAtRight = thrust::get<2>(t);
		int blocAtTop = thrust::get<3>(t);
		int blocAtBottom = thrust::get<4>(t);
		int blocAtFront = thrust::get<5>(t);
		int blocAtBack = thrust::get<6>(t);
		int voisins = 0;
		
		bool all = true; //verifie s'il y a une condition
		if (!isOnLeftBorder(index))
			if (all || blocAtLeft == filtre)
				voisins++;
		if (!isOnRightBorder(index))
			if (all || blocAtRight == filtre)
				voisins++;
		if (!isOnTopBorder(index))
			if (all || blocAtTop == filtre)
				voisins++;
		if (!isOnBottomBorder(index))
			if (all || blocAtBottom == filtre)
				voisins++;
		if (!isOnFrontBorder(index))
			if (all || blocAtFront == filtre)
				voisins++;
		if (!isOnBackBorder(index))
			if (all || blocAtBack == filtre)
				voisins++;
		thrust::get<7>(t) = voisins;
	}
};



/*
int indexFourmiVoisine(int index, , vector<int> &matTransitions) {
	if (!isOnRightBorder(index))
		if (matTransitions[index+1] == index || matTransitions[index+1] == -1*index-2)
			return index+1;
	if (!isOnLeftBorder(index))
		if (matTransitions[index-1] == index || matTransitions[index-1] == -1*index-2)
			return index-1;
	if (!isOnTopBorder(index))
		if (matTransitions[index-taille] == index || matTransitions[index-taille] == -1*index-2)
			return index-taille;
	if (!isOnBottomBorder(index))
		if (matTransitions[index+taille] == index || matTransitions[index+taille] == -1*index-2)
			return index+taille;
	if (!isOnFrontBorder(index)) // A reformuler
		if (matTransitions[index+taille**2] == index || matTransitions[index+taille**2] == -1*index-2)
			return index+taille**2;
	if (!isOnBackBorder(index)) // A reformuler
		if (matTransitions[index-taille**2] == index || matTransitions[index-taille**2] == -1*index-2)
			return index-taille**2;
	return -1;
}*/


struct genereMatrix {
	__host__ __device__
	int operator()(int bloc) {
		
		int states[] = {ACCESSIBLE, GRAIN};
		
		int x = bloc %2;
		return states[x];
	}
};

struct placeAnt {
	const int a;

	placeAnt(int _a) : a(_a) {}
	
	__host__ __device__
	int operator()(int bloc) {
		if (bloc == ACCESSIBLE)
			
			//if (a%6 < 2 && getNbFourmi() < 1) {
				//nbFourmi += 1;
				return FOURMI;
			//}
		return bloc;
	}
};



struct updateStates2 {
	
	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t) {
		/*t(0) = matfourmi
		t(1) = voisins actifs
		t(2) = isaccessible
		t(3) = matfourmi
		*/
		int bloc = thrust::get<0>(t);
		int nbVoisinsActifs = thrust::get<1>(t);
		bool isAccessible = thrust::get<2>(t);
		
		if (bloc == VIDE || bloc == ACCESSIBLE) {
			if (isAccessible && nbVoisinsActifs<=1)
				thrust::get<3>(t) = ACCESSIBLE;
			else
				thrust::get<3>(t) = VIDE;
		}
		else if (bloc == GRAIN && nbVoisinsActifs>1)
			thrust::get<3>(t) = GRAIN_CONFLIT;
		else if (bloc == GRAIN_CONFLIT && nbVoisinsActifs<=1)
			thrust::get<3>(t) = GRAIN;
		else
			thrust::get<3>(t) = bloc;
		thrust::get<3>(t) = -1;
	}
	
};

thrust::host_vector<int> updateStates (thrust::host_vector<int> &matFourmi) {
	
	int tailleTotale = matFourmi.size();
	
	// Cr�ation des matrices d�cal�es
	thrust::counting_iterator<int> begin(0);
	thrust::counting_iterator<int> end(tailleTotale);

	thrust::host_vector <int> rightIndexes(tailleTotale);
	thrust::host_vector <int> leftIndexes(tailleTotale);
	thrust::host_vector <int> topIndexes(tailleTotale);
	thrust::host_vector <int> bottomIndexes(tailleTotale);
	thrust::host_vector <int> frontIndexes(tailleTotale);
	thrust::host_vector <int> backIndexes(tailleTotale);
	 
	thrust::transform(begin, end, leftIndexes.begin(), moveIndex(-1 ,tailleTotale));
	thrust::transform(begin, end, rightIndexes.begin(), moveIndex(1 ,tailleTotale));
	thrust::transform(begin, end, topIndexes.begin(), moveIndex(-taille ,tailleTotale));
	thrust::transform(begin, end, bottomIndexes.begin(), moveIndex(taille ,tailleTotale));
	thrust::transform(begin, end, frontIndexes.begin(), moveIndex(taille*taille ,tailleTotale));
	thrust::transform(begin, end, backIndexes.begin(), moveIndex(-taille*taille ,tailleTotale));

	
	// Initialisation de la matrice des bool�ens accessibles
	thrust::host_vector<int> matIsAccessible;
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				begin, 
				thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.begin()), 
				matIsAccessible.begin()
			)
		),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				end, 
				thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.end()), 
				matIsAccessible.end()
			)
		),
		isAccessible()
	);
	
	
	// Initialisation de la matrice des voisins actifs
	thrust::host_vector<thrust::host_vector<int>> matVoisinsActifs;
	thrust::host_vector<int> matNbVoisinsActifs;
	
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				begin, 
				thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.begin()), 
				matNbVoisinsActifs.begin()
			)
		),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				end, 
				thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.end()), 
				matNbVoisinsActifs.end()
			)
		),
		listeNbVoisinsActifs()
	);
	
		
		
	// Application des conditions d'updateStates2 sur les 3 matrices transform�es en tuple (la fonction transform ne prend que 2 elements max)
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(matFourmi.begin(), matNbVoisinsActifs.begin(), matIsAccessible.begin(), matFourmi.begin())
		),
		thrust::make_zip_iterator(
			thrust::make_tuple(matFourmi.end(), matNbVoisinsActifs.end(), matIsAccessible.end(), matFourmi.end())
		),
		updateStates2()
	);
	
	return matFourmi;
}

/*
// index : position dans la matrice
// bloc ; �tat du bloc � la position "index"
int transition(int index, int bloc) {
	int choix = rand() % 2;
	if (bloc==FOURMI || bloc==TRANSIT) {
		if (choix==0) { //D�placement
			voisins = listeVoisinsAccessibles(index);
			return deplacement_alea(voisins);
		}
		else if (choix==1 && bloc == FOURMI) { //Ramassage
			vector <int> tmp;
			tmp.push_back(GRAIN);
			vector <int> voisins = listeVoisins(index, tmp);
			return deplacement_alea(voisins);
		}
		else if (choix==1 && bloc == TRANSIT) { //D�pot
			vector <int> voisins = listeVoisinsAccessibles(index);
			if (deplacement_alea(voisins)==-1)
				return -1;
			else:
				return -1*deplacement_alea(voisins)-2;
		}
	}
	else
		return -1;
}

int transition2(int index) {
	int val = matTransitions[index];
	bool isDeparture = val != -1;
	int indexFourmi = indexFourmiVoisine(index);
	bool isArrival = indexFourmi != -1;

	if (isDeparture) {
		if (val > -1) //cas d�placement
			return ACCESSIBLE;
		else	//cas d�pot
			return GRAIN;
	}
	else if (isArrival) {
		if (matTransitions[indexFourmi] > -1) { //cas d�placement
			if (matfourmi[index] == ACCESSIBLE) //cas d�placement simple
				return matfourmi[indexFourmi]; 
			else if (matfourmi[index] == GRAIN) //cas ramassage
				return TRANSIT;
		}
		else if (matTransitions[indexFourmi] < -1) {	//cas d�pot
			return FOURMI;
		else
			cout << "ERREUR DE MERDE" << endl;
	}
	else
		return matfourmi[index];
}*/


int main() {
	
	srand ( time(NULL) );
	
	thrust::host_vector<int> matFourmi(taille*taille*taille);

	clock_t t1;
	clock_t t2;
	t1 = clock();

	t2=clock()-t1;
	t1 = clock();
	
	// G�n�ration de la matrice
	thrust::generate(matFourmi.begin(), matFourmi.end(), rand);
	thrust::transform(matFourmi.begin(), matFourmi.end(), matFourmi.begin(), genereMatrix());
	
	// Placement d'une fourmi -------- A modifier : faire une boucle pour plusieurs fourmis
	int randvalue = rand() % taille*taille*taille;
	matFourmi[randvalue] = FOURMI;
	
	// Mise � jour de la matrice
	matFourmi = updateStates(matFourmi);
	
	t2 = clock() - t1;
	
	cout << "Temps �coul� : " << t2 << endl;

	for(int i = 0; i < taille*taille*taille; i++) {
		std::cout<< matFourmi[i] << std::endl;
	}
	return 0;
}

/*
int main() {

	int tailleMatrice = 3;
	int nbEtapes = 0;

	// A reprendre du fichier de tests

	cout << "Matrice initiale" << endl << matfourmi << endl;
	cout << "Combien d'etapes voulez vous realiser ?" << endl;
	cin >> nbEtapes;

	for (int i=0 ; i<nbEtapes ; i++) {
		cout << "Etape " << i << endl;
	}

}*/

/*
nvcc --machine 32 -ccbin "C:\Program Files\Microsoft Visual Studio 10.0\VC\bin"  -I "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v4.0\include" testpara.cu -o testpara
*/