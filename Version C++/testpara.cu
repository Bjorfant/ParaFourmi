#include "hip/hip_runtime.h"
#include <thrust/for_each.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <iostream>
#include <vector>
#include <cstdlib>

#define taille 3

#define GET_LEFT(n) (n-1)
#define GET_RIGHT(n) (n+1)
#define GET_TOP(n) (n-taille)
#define GET_BOTTOM(n) (n+taille)
#define GET_FRONT(n) (n+taille*taille)
#define GET_BACK(n) (n-taille*taille)



using namespace std;
/*********************************************
MODELE DE DONNEES
*********************************************/

 typedef enum {
	VIDE=0,			//une case qui est complètement isolée des blocs pleins
	ACCESSIBLE=1,	//une case qui a au moins un voisin grain
	ACCESSIBLE_CONFLIT=2,	//une case qui est accessible et a au moins deux voisins actifs
	GRAIN_CONFLIT=3,	//une case grain qui a au moins deux voisin actifs
	GRAIN=4,		//une case de terre
	FOURMI=5,		//une fourmi
	TRANSIT=6		//une fourmi transportant un bloc
} State;


/*********************************************
FONCTIONS UTILES
*********************************************/

void printMatrix(thrust::host_vector<int> matFourmi) {
	for(int i = 0; i < taille*taille*taille; i++) {
		cout<< matFourmi[i];
		if (i%taille==taille-1)
			cout << endl;
		if (i%(taille*taille) == taille*taille-1)
			cout << endl;
	}
}

//retourne vrai si la case d'indice "index" est sur le bord gauche de la matrice
__host__ __device__
int isOnLeftBorder(int index) {
	return index%taille == 0;
}

//retourne vrai si la case d'indice "index" est sur le bord droit de la matrice 
__host__ __device__
int isOnRightBorder(int index) {
	return index%taille == taille - 1;
}

//retourne vrai si la case d'indice "index" est sur le bord supérieur de la matrice
__host__ __device__
int isOnTopBorder(int index) {
	return index%(taille*taille) - taille < 0;
}

//retourne vrai si la case d'indice "index" est sur le bord inférieur de la matrice
__host__ __device__
int isOnBottomBorder(int index) {
	return index%(taille*taille) + taille >= taille*taille;
}

//retourne vrai si la case d'indice "index" est sur le bord avant de la matrice 
__host__ __device__
int isOnFrontBorder(int index) {
	return index + (taille*taille) >= taille*taille*taille;
}

//retourne vrai si la case d'indice "index" est sur le bord arrière de la matrice
__host__ __device__
int isOnBackBorder(int index) {
	return index < taille*taille;
}


//décale un indice d'un vecteur d'une certaine valeur. 
//Permet de créer un vecteur de mappage pour décaller un vecteur de façon cyclique
struct moveIndex {
	const int delta, maxIndex;

	//la stucture prend le décalage à appliquer ainsi que l'indice maximum du vecteur
	moveIndex(int _delta, int _maxIndex) : delta(_delta), maxIndex(_maxIndex) {}

	__host__ __device__
	int operator()(int index){
		int val = (index + delta)%maxIndex;
		if (val<0)
			return val+maxIndex;
		else
			return val;
	}
};

__host__ __device__
int destination_alea(int index, int blocAtLeft, int blocAtRight, int blocAtTop, int blocAtBottom, int blocAtFront, int blocAtBack, int type) {
	if (!isOnLeftBorder(index)) { 
		if (blocAtLeft==type)
			return GET_LEFT(index);
	}
	if (!isOnRightBorder(index)) {
		if (blocAtRight==type)
			return GET_RIGHT(index);
	}
	if (!isOnTopBorder(index)) {
		if (blocAtLeft==type)
			return GET_TOP(index);
	}
	if (!isOnBottomBorder(index)) {
		if (blocAtLeft==type)
			return GET_BOTTOM(index);
	}
	if (!isOnFrontBorder(index)) {
		if (blocAtLeft==type)
			return GET_FRONT(index);
	}
	if (!isOnBackBorder(index)) {
		if (blocAtLeft==type)
			return GET_BACK(index);
	}
	return -1;
}


__host__ __device__
int indexFourmiArrivante(int index,
			int matTransitionsBlocAtLeft,
			int matTransitionsBlocAtRight,
			int matTransitionsBlocAtTop, 
			int matTransitionsBlocAtBottom,
			int matTransitionsBlocAtFront,
			int matTransitionsBlocAtBack
) {
	if (!isOnRightBorder(index))
		if (matTransitionsBlocAtRight == index || matTransitionsBlocAtRight == -1*index-2)
			return GET_RIGHT(index);
	if (!isOnLeftBorder(index))
		if (matTransitionsBlocAtLeft == index || matTransitionsBlocAtLeft == -1*index-2)
			return GET_LEFT(index);
	if (!isOnTopBorder(index))
		if (matTransitionsBlocAtTop == index || matTransitionsBlocAtTop == -1*index-2)
			return GET_TOP(index);
	if (!isOnBottomBorder(index))
		if (matTransitionsBlocAtBottom == index || matTransitionsBlocAtBottom == -1*index-2)
			return GET_BOTTOM(index);
	if (!isOnFrontBorder(index)) // A reformuler
		if (matTransitionsBlocAtFront == index || matTransitionsBlocAtFront == -1*index-2)
			return GET_FRONT(index);
	if (!isOnBackBorder(index)) // A reformuler
		if (matTransitionsBlocAtBack == index || matTransitionsBlocAtBack == -1*index-2)
			return GET_BACK(index);
	return -1;
}


//genère une matrice aléatoire constituée de grains et de bloc accessibles 
//cette matrice nécéssite d'être retravaillée par la suite pour la cohérence des données
struct genereMatrix {
	__host__ __device__
	int operator()(int bloc) {
		
		int states[] = {ACCESSIBLE, GRAIN};
		
		int x = bloc %2;
		return states[x];
	}
};


//retourne le nombre de voisins actifs dans l'entourage de la case à la position "index" dans la matrice
//un voisin actif est une fourmi simple ou une fourmi transportant un bloc
__host__ __device__
int getNbVoisinsActifs(int index, int left, int right, int top, int bottom, int front, int back) {
	int nb = 0;
	if (!isOnLeftBorder(index)) {
		if (left == FOURMI || left == TRANSIT) 
			nb++;
	}
	if (!isOnRightBorder(index)) {
		if (right == FOURMI || right == TRANSIT) 
			nb++;
	}
	if (!isOnTopBorder(index)) {
		if (top == FOURMI || top == TRANSIT) 
			nb++;
	}
	if (!isOnBottomBorder(index)) {
		if (bottom == FOURMI || bottom == TRANSIT) 
			nb++;
	}
	if (!isOnFrontBorder(index)) {
		if (front == FOURMI || front == TRANSIT) 
			nb++;
	}
	if(!isOnBackBorder(index)) {
		if (back == FOURMI || back == TRANSIT) 
			nb++;
	} 
	return nb;
}

//retourne vrai si la case d'indice "index" est accessible 
__host__ __device__
bool isAccessible(int index, int left, int right, int top, int bottom, int front, int back) {
	
	bool isAccessible = false;
	
	if (!isOnLeftBorder(index)) { 
		isAccessible |= left == GRAIN || left == GRAIN_CONFLIT;
	}
	if (!isOnRightBorder(index)) {
		isAccessible |= right == GRAIN || right == GRAIN_CONFLIT;
	}
	if (!isOnTopBorder(index)) {
		isAccessible |= top == GRAIN || top == GRAIN_CONFLIT;
	}
	if (!isOnBottomBorder(index)) {
		isAccessible |= bottom == GRAIN || bottom == GRAIN_CONFLIT;
	}
	if (!isOnFrontBorder(index)) {
		isAccessible |= front == GRAIN || front == GRAIN_CONFLIT;
	}
	if (!isOnBackBorder(index)) {
		isAccessible |= back == GRAIN || back == GRAIN_CONFLIT;
	}
	return isAccessible;
}



/*********************************************
FONCTIONS PRINCIPALES DES BOUCLES DE SIMULATION
*********************************************/

//la fonction updateStates sert à garder l'intégrité des données présentes dans la matrice
//on vérifie ici que tous les états sont cohérents vis à vis du modèle adopté
//elle prend en paramètres un tuple de 8 valeurs dont :
//- l'indice auquel on se trouve dans la matrice
//- la valeur du bloc courant 
//- les 6 valeurs des des blocs voisins
struct updateStates {

template <typename Tuple>
__host__ __device__
	void operator()(Tuple t) {
		int index = thrust::get<0>(t);
		int bloc = thrust::get<1>(t);
		int blocAtLeft = thrust::get<2>(t);
		int blocAtRight = thrust::get<3>(t);
		int blocAtTop = thrust::get<4>(t);
		int blocAtBottom = thrust::get<5>(t);
		int blocAtFront = thrust::get<6>(t);
		int blocAtBack = thrust::get<7>(t);

		//dans le cas ou le bloc courant est vide ou accessible on vérifie que ses prorpiétés correspondent bien à son état
		if(bloc == ACCESSIBLE || bloc == ACCESSIBLE_CONFLIT) {
			if(!isAccessible(index, blocAtLeft, blocAtRight, blocAtTop, blocAtBottom, blocAtFront, blocAtBack))
				thrust::get<8>(t) = VIDE; 
			else {
				if(getNbVoisinsActifs(index, blocAtLeft, blocAtRight, blocAtTop, blocAtBottom, blocAtFront, blocAtBack) > 1)
					thrust::get<8>(t) = ACCESSIBLE_CONFLIT;
				else
					thrust::get<8>(t) = ACCESSIBLE;
			}
		} else if (bloc == GRAIN || bloc == GRAIN_CONFLIT) {
			//Dans le cas ou le bloc est un grain ou du plein il faut vérifier que ses propriétés correspondent bien à son état 
			if (getNbVoisinsActifs(index, blocAtLeft, blocAtRight, blocAtTop, blocAtBottom, blocAtFront, blocAtBack) > 1)
				thrust::get<8>(t) = GRAIN_CONFLIT;
			else
				thrust::get<8>(t) = GRAIN;
		} else {
			//si le bloc est de type fourmi ou transit alors il ne change pas d'état 
			//on retourne sa valeur inchangée
			thrust::get<8>(t) = bloc;
		}
	}
	};


/*Cette fonction permet de determiner les intention d'action de toutes les fourmis présentes dans la matrice.
Les actions ne sont pas effectuées sur la matrice principale, elles sont simplement renseigner.
On créé ainsi une matrice supplémentaire indiquant les mouvements qui interviendront à l'étape suivante.
*/
struct transition1 {

	template <typename Tuple>
	__host__ __device__
	int operator() (Tuple t) {
	
		int index = thrust::get<0>(t);
		int bloc = thrust::get<1>(t);
		int blocAtLeft = thrust::get<2>(t);
		int blocAtRight = thrust::get<3>(t);
		int blocAtTop = thrust::get<4>(t);
		int blocAtBottom = thrust::get<5>(t);
		int blocAtFront = thrust::get<6>(t);
		int blocAtBack = thrust::get<7>(t);
		
		int choix = 0;
		if (bloc==FOURMI || bloc==TRANSIT) {
			if (choix==0) { //Déplacement
				return destination_alea(index,blocAtLeft,blocAtRight,blocAtTop,blocAtBottom,blocAtFront,blocAtBack, ACCESSIBLE);
			}
			else if (choix==1 && bloc == FOURMI) { //Ramassage
				//vector <int> tmp;
				//tmp.push_back(GRAIN);
				//vector <int> voisins = listeVoisins(index, tmp);
				return destination_alea(index,blocAtLeft,blocAtRight,blocAtTop,blocAtBottom,blocAtFront,blocAtBack, GRAIN);
			}
			else if (choix==1 && bloc == TRANSIT) { //Dépot
				if (destination_alea(index,blocAtLeft,blocAtRight,blocAtTop,blocAtBottom,blocAtFront,blocAtBack, ACCESSIBLE)==-1)
					return -1;
				else
					return -1*destination_alea(index,blocAtLeft,blocAtRight,blocAtTop,blocAtBottom,blocAtFront,blocAtBack, ACCESSIBLE)-2;
			}
		}
		else
			return -1;
		return -1;
	}
};


struct transition2 {

	template <typename Tuple1,typename Tuple2>
	__host__ __device__
	int operator() (Tuple1 t1, Tuple2 t2) {
		
		int index = thrust::get<0>(t1);
		int blocOriginal = thrust::get<1>(t1); // matFourmi[index]
		
		
		//int blocTransitions = thrust::get<1>(t); //matTransitions[indexFourmi]
		//int blocArrivee =  thrust::get<2>(t); // matFourmi[indexFourmi]
		
		int matFourmiBlocAtLeft = thrust::get<2>(t1);
		int matFourmiBlocAtRight = thrust::get<3>(t1);
		int matFourmiBlocAtTop = thrust::get<4>(t1);
		int matFourmiBlocAtBottom = thrust::get<5>(t1);
		int matFourmiBlocAtFront = thrust::get<6>(t1);
		int matFourmiBlocAtBack = thrust::get<7>(t1);
		
		int blocTransition = thrust::get<0>(t2);
		int matTransitionsBlocAtLeft = thrust::get<1>(t2);
		int matTransitionsBlocAtRight = thrust::get<2>(t2);
		int matTransitionsBlocAtTop = thrust::get<3>(t2);
		int matTransitionsBlocAtBottom = thrust::get<4>(t2);
		int matTransitionsBlocAtFront = thrust::get<5>(t2);
		int matTransitionsBlocAtBack = thrust::get<6>(t2);
	
		//récupère, si elle existe, la position de la fourmi qui arrive sur la case courante
		int indexFourmi = indexFourmiArrivante(index, matTransitionsBlocAtLeft, matTransitionsBlocAtRight, matTransitionsBlocAtTop, matTransitionsBlocAtBottom, matTransitionsBlocAtFront, matTransitionsBlocAtBack);
		
		bool isDeparture = blocTransition != -1;
		bool isArrival = indexFourmi != -1;
		
		int blocTransitions = 0;
		int blocArrivee = 0;
		
		if (indexFourmi == GET_LEFT(index)) {
			blocTransitions = matTransitionsBlocAtLeft;
			blocArrivee = matFourmiBlocAtLeft;
		}
		else if (indexFourmi == GET_RIGHT(index)) {
			blocTransitions = matTransitionsBlocAtRight;
			blocArrivee = matFourmiBlocAtRight;
		}
		else if (indexFourmi == GET_TOP(index)) {
			blocTransitions = matTransitionsBlocAtTop;
			blocArrivee = matFourmiBlocAtTop;
		}
		else if (indexFourmi == GET_BOTTOM(index)) {
			blocTransitions = matTransitionsBlocAtBottom;
			blocArrivee = matFourmiBlocAtBottom;
		}
		else if (indexFourmi == GET_FRONT(index)) {
			blocTransitions = matTransitionsBlocAtFront;
			blocArrivee = matFourmiBlocAtFront;
		}
		else if (indexFourmi == GET_BACK(index)) {
			blocTransitions = matTransitionsBlocAtBack;
			blocArrivee = matFourmiBlocAtBack;
		}

		if (isDeparture) {
			if (blocOriginal > -1) //cas déplacement
				return ACCESSIBLE;
			else	//cas dépot
				return GRAIN;
		}
		else if (isArrival) {
			if (blocTransitions > -1) { //cas déplacement
				if (blocOriginal == ACCESSIBLE) //cas déplacement simple
					return blocArrivee; 
				else if (blocOriginal == GRAIN) //cas ramassage
					return TRANSIT;
			}
			else if (blocTransitions < -1)	//cas dépot
				return FOURMI;
		}
		else
			return blocOriginal;
		
		return 1;
	}
};


int main() {
	
	srand ( time(NULL) );
	clock_t t1;
	clock_t t2;
	t1 = clock();
	
	
	// Génération de la matrice
	thrust::host_vector<int> matFourmi(taille*taille*taille);
	thrust::generate(matFourmi.begin(), matFourmi.end(), rand);
	thrust::transform(matFourmi.begin(), matFourmi.end(), matFourmi.begin(), genereMatrix());
	
	// Placement d'une fourmi
	int nbFourmis = 1;
	for (int i = 0 ; i<nbFourmis ; i++) {
		int randvalue = rand() % taille*taille*taille;
		matFourmi[randvalue] = FOURMI;
	}
	

	// Création des matrices décalées
	int tailleTotale = matFourmi.size();
	thrust::counting_iterator<int> begin(0);
	thrust::counting_iterator<int> end(tailleTotale);

	thrust::host_vector <int> rightIndexes(tailleTotale);
	thrust::host_vector <int> leftIndexes(tailleTotale);
	thrust::host_vector <int> topIndexes(tailleTotale);
	thrust::host_vector <int> bottomIndexes(tailleTotale);
	thrust::host_vector <int> frontIndexes(tailleTotale);
	thrust::host_vector <int> backIndexes(tailleTotale);
	
	//création des vecteurs contenant les indices décalés du vecteur principal
	thrust::transform(begin, end, leftIndexes.begin(), moveIndex(-1 ,tailleTotale));
	thrust::transform(begin, end, rightIndexes.begin(), moveIndex(1 ,tailleTotale));
	thrust::transform(begin, end, topIndexes.begin(), moveIndex(-taille ,tailleTotale));
	thrust::transform(begin, end, bottomIndexes.begin(), moveIndex(taille ,tailleTotale));
	thrust::transform(begin, end, frontIndexes.begin(), moveIndex(taille*taille ,tailleTotale));
	thrust::transform(begin, end, backIndexes.begin(), moveIndex(-taille*taille ,tailleTotale));
	
	
	//Première mise à jour de la matrice
	//ici, on créé des listes décalées pour pouvoir acceder à tous les éléments voisins d'un élément particulier
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				begin,
				matFourmi.begin(),
				thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.begin()), 
				matFourmi.begin()
			)
		),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				end,
				matFourmi.end(), 
				thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.end()), 
				matFourmi.end()
			)
		),
		updateStates()
	);
	
	// Création de la matrice intermédiaire
	thrust::host_vector <int> matTransitions(tailleTotale);
	thrust::fill(matTransitions.begin(), matTransitions.end(), 0);

	cout << "Matrice initiale" << endl;
	printMatrix(matFourmi);
	
	// Demande du nombre d'étapes à l'utilisateur
	int nbEtapes;
	cout << "Combien d'etapes voulez vous realiser ?" << endl;
	cin >> nbEtapes;
	
	
	// Boucle principale des étapes
	for (int i=0 ; i<nbEtapes ; i++) {
		cout << "Etape " << i << endl;
		
		// Transition 1
		thrust::transform(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					begin,
					matFourmi.begin(),
					thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.begin())
				)
			),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					end,
					matFourmi.end(), 
					thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.end())
				)
			),
			matTransitions.begin(),
			transition1()
		);
		cout << "\nMatrice temporaire" << endl;
		printMatrix(matTransitions);
		
		// Transition 2
		
		thrust::transform(
			thrust::make_zip_iterator(
					thrust::make_tuple(
						begin,
						matFourmi.begin(),
						thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.begin()), 
						thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.begin()), 
						thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.begin()), 
						thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.begin()), 
						thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.begin()), 
						thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.begin())
					)
			),
			thrust::make_zip_iterator(
					thrust::make_tuple(
						end,
						matFourmi.end(),
						thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.end()), 
						thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.end()), 
						thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.end()), 
						thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.end()), 
						thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.end()), 
						thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.end())
					)
			),
			thrust::make_zip_iterator(
					thrust::make_tuple(
						matTransitions.begin(),
						thrust::make_permutation_iterator(matTransitions.begin(), leftIndexes.begin()), 
						thrust::make_permutation_iterator(matTransitions.begin(), rightIndexes.begin()), 
						thrust::make_permutation_iterator(matTransitions.begin(), topIndexes.begin()), 
						thrust::make_permutation_iterator(matTransitions.begin(), bottomIndexes.begin()), 
						thrust::make_permutation_iterator(matTransitions.begin(), frontIndexes.begin()), 
						thrust::make_permutation_iterator(matTransitions.begin(), backIndexes.begin())
					)
			),
			matFourmi.begin(),
			transition2()
		);
		
		//matFourmi = updateStatesHost(matFourmi);
		printMatrix(matFourmi);
		
		system("pause");
	}
	
	t2 = clock() - t1;
	cout << "Temps écoulé : " << t2 << endl;

	return 0;
}

/*
nvcc --machine 32 -ccbin "C:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin"  -I "C:\Program Files (x86)\NVIDIA GPU Computing Toolkit\CUDA\v3.2\include" testpara.cu -o testpara
*/