template Type randomchoice(vector<Type> v) {
	int random = rand() % v.size();
	return v[random];
}

//retourne vrai si la case d'indice "index" est sur le bord gauche de la matrice
int isOnLeftBorder(index) {
	return index%taille == 0;
}

//retourne vrai si la case d'indice "index" est sur le bord droit de la matrice 
int isOnRightBorder(index) {
	return index%taille == taille - 1;
}

//retourne vrai si la case d'indice "index" est sur le bord sup�rieur de la matrice
int isOnTopBorder(index) {
	return index%(taille**2) - taille < 0;
}

//retourne vrai si la case d'indice "index" est sur le bord inf�rieur de la matrice
int isOnBottomBorder(index) {
	return index%(taille**2) + taille >= taille**2;
}

//retourne vrai si la case d'indice "index" est sur le bord avant de la matrice 
int isOnFrontBorder(index) {
	return index + (taille**2) >= taille**3;
}

//retourne vrai si la case d'indice "index" est sur le bord arri�re de la matrice
int isOnBackBorder(index) {
	return index < taille**2;
}

bool isAccessible(index) {
	# Calcul du nombre de voisins
	if (!isOnRightBorder(index)) {
		if (matfourmi[index+1] == State::GRAIN)
			return true;
	}
	if (!isOnLeftBorder(index)) {
		if (matfourmi[index-1] == State::GRAIN)
			return true;
	}
	if (!isOnTopBorder(index)) {
		if matfourmi[index-taille] == State::GRAIN)
			return true;
	}
	if (!isOnBottomBorder(index)) {
		if (matfourmi[index+taille] == State::GRAIN)
			return true;
	}
	if (!isOnFrontBorder(index)) {
		if (matfourmi[index+taille**2] == State::GRAIN)
			return true;
	}
	if (!isOnBackBorder(index)) {
		if (matfourmi[index-taille**2] == State::GRAIN)
			return true;
	}
	return false;
}

int deplacement_alea(voisins) {
	if voisins.size() >= 1
		return randomchoice(voisins);
	else:
		return -1;
}

//r�cup�re les voisins d'une case de la matrice
//possibilit� de filtrer les voisins par une liste d'�tat que l'on cherche 
//si la liste est vide on renvoit tous les voisins
vector <int> listeVoisins(int index, vector <int> filtre) {
	voisins=[]
	all = filtre == [] //verifie si il a une condition
	if (!isOnRightBorder(index))
		if (all || matfourmi[index+1] in filtre)
			voisins.push_back(index+1);
	if (!isOnLeftBorder(index))
		if (all || matfourmi[index-1] in filtre)
			voisins.push_back(index-1);
	if (!isOnTopBorder(index))
		if (all || matfourmi[index-taille] in filtre)
			voisins.push_back(index-taille);
	if (!isOnBottomBorder(index))
		if (all || matfourmi[index+taille] in filtre)
			voisins.push_back(index+taille);
	if (!isOnFrontBorder(index))
		if (all || matfourmi[index+taille**2] in filtre)
			voisins.push_back(index+taille**2);
	if (!isOnBackBorder(index))
		if (all || matfourmi[index-taille**2] in filtre)
			voisins.push_back(index-taille**2);
	return voisins;
}

vector <int> listeVoisinsAccessibles(int index) {
	vector <int> v;
	v.push_back(State::ACCESSIBLE);
	return listeVoisins(index, v);
}
	
vector <int> listeVoisinsActifs(int index) {
	vector <int> v;
	v.push_back(State::ACCESSIBLE);
	v.push_back(State::TRANSIT);
	return listeVoisins(index, v);
	
	
int indexFourmiVoisine(int index) {
	if (!isOnRightBorder(index))
		if (matTransitions[index+1] == index || matTransitions[index+1] == -1*index-2)
			return index+1;
	if (!isOnLeftBorder(index))
		if (matTransitions[index-1] == index || matTransitions[index-1] == -1*index-2)
			return index-1;
	if (!isOnTopBorder(index))
		if (matTransitions[index-taille] == index || matTransitions[index-taille] == -1*index-2)
			return index-taille;
	if (!isOnBottomBorder(index))
		if (matTransitions[index+taille] == index || matTransitions[index+taille] == -1*index-2)
			return index+taille;
	if (!isOnFrontBorder(index)) // A reformuler
		if (matTransitions[index+taille**2] == index || matTransitions[index+taille**2] == -1*index-2)
			return index+taille**2;
	if (!isOnBackBorder(index)) // A reformuler
		if (matTransitions[index-taille**2] == index || matTransitions[index-taille**2] == -1*index-2)
			return index-taille**2;
	return -1;
}
	
	
// index : position dans la matrice
// bloc ; �tat du bloc � la position "index"
int transition(int index, int bloc):
	int choix = rand() % 2;
	if (bloc==State::FOURMI || bloc==State::TRANSIT) {
		if (choix==0) { //D�placement
			voisins = listeVoisinsAccessibles(index);
			return deplacement_alea(voisins);
		}
		else if (choix==1 && bloc == State::FOURMI) { //Ramassage
			vector <int> tmp;
			tmp.push_back(State::GRAIN);
			vector <int> voisins = listeVoisins(index, tmp);
			return deplacement_alea(voisins);
		}
		else if (choix==1 && bloc == State::TRANSIT { //D�pot
			vector <int> voisins = listeVoisinsAccessibles(index);
			if (deplacement_alea(voisins)==-1)
				return -1;
			else:
				return -1*deplacement_alea(voisins)-2;
		}
	}
	else
		return -1;
}

int transition2(int index) {
	int val = matTransitions[index];
	bool isDeparture = val != -1;
	int indexFourmi = indexFourmiVoisine(index);
	bool isArrival = indexFourmi != -1;
	
	if (isDeparture) {
		if (val > -1) //cas d�placement
			return State::ACCESSIBLE;
		else	//cas d�pot
			return State::GRAIN;
	}
	else if (isArrival) {
		if (matTransitions[indexFourmi] > -1) { //cas d�placement
			if (matfourmi[index] == State::ACCESSIBLE) //cas d�placement simple
				return matfourmi[indexFourmi]; 
			else if (matfourmi[index] == State::GRAIN) //cas ramassage
				return State::TRANSIT;
		}
		else if (matTransitions[indexFourmi] < -1) {	//cas d�pot
			return State::FOURMI;
		else
			cout << "ERREUR DE MERDE" << endl;
	}
	else
		return matfourmi[index];
}

vector <int> updateStates(int index, int bloc) {
	int nbVoisinsActifs = listeVoisinsActifs(index).size();
	if (bloc == State::VIDE || bloc == State::ACCESSIBLE) {
		if (isAccessible(index) && nbVoisinsActifs<=1)
			return State::ACCESSIBLE;
		else
			return State::VIDE;
	}
	else if (bloc == State::GRAIN && nbVoisinsActifs>1)
		return State::GRAIN_CONFLIT;
	else if (bloc == State::GRAIN_CONFLIT && nbVoisinsActifs<=1)
		return State::GRAIN;
	else:	
		return bloc;
}

int main() {

	int tailleMatrice = 3;
	int nbEtapes = 0;
	
	// A reprendre du fichier de tests
	
	cout << "Matrice initiale" << endl << matfourmi << endl;
	cout << "Combien d'etapes voulez vous realiser ?" << endl;
	cin >> nbEtapes;
	
	for (int i=0 ; i<nbEtapes ; i++) {
		cout << "Etape " << i << endl;
	}

}