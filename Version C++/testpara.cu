#include "hip/hip_runtime.h"
#include <thrust/for_each.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <iostream>
#include <vector>
#include <cstdlib>

#define taille 3

using namespace std;
/*********************************************
MODELE DE DONNEES
*********************************************/

 typedef enum {
	VIDE=0,			//une case qui est compl�tement isol�e des blocs pleins
	ACCESSIBLE=1,	//une case qui a au moins un voisin grain
	ACCESSIBLE_CONFLIT=2,	//une case qui est accessible et a au moins deux voisins actifs
	GRAIN_CONFLIT=3,	//une case grain qui a au moins deux voisin actifs
	GRAIN=4,		//une case de terre
	FOURMI=5,		//une fourmi
	TRANSIT=6		//une fourmi transportant un bloc
} State;


/*********************************************
FONCTIONS UTILES
*********************************************/

void printMatrix(thrust::host_vector<int> matFourmi) {
	for(int i = 0; i < taille*taille*taille; i++) {
		cout<< matFourmi[i] << endl;
	}
}

//retourne vrai si la case d'indice "index" est sur le bord gauche de la matrice

int isOnLeftBorder(int index) {
	return index%taille == 0;
}

//retourne vrai si la case d'indice "index" est sur le bord droit de la matrice 

int isOnRightBorder(int index) {
	return index%taille == taille - 1;
}

//retourne vrai si la case d'indice "index" est sur le bord sup�rieur de la matrice

int isOnTopBorder(int index) {
	return index%(taille*taille) - taille < 0;
}

//retourne vrai si la case d'indice "index" est sur le bord inf�rieur de la matrice

int isOnBottomBorder(int index) {
	return index%(taille*taille) + taille >= taille*taille;
}

//retourne vrai si la case d'indice "index" est sur le bord avant de la matrice 

int isOnFrontBorder(int index) {
	return index + (taille*taille) >= taille*taille*taille;
}

//retourne vrai si la case d'indice "index" est sur le bord arri�re de la matrice

int isOnBackBorder(int index) {
	return index < taille*taille;
}


//d�cale un indice d'un vecteur d'une certaine valeur. 
//Permet de cr�er un vecteur de mappage pour d�caller un vecteur de fa�on cyclique
struct moveIndex {
	const int delta, maxIndex;

	//la stucture prend le d�calage � appliquer ainsi que l'indice maximum du vecteur
	moveIndex(int _delta, int _maxIndex) : delta(_delta), maxIndex(_maxIndex) {}

	__host__ __device__
	int operator()(int index){
		return (index + delta)%maxIndex;
	}
};


int deplacement_alea(vector <int> voisins) {
	if (voisins.size() >= 1)
		return voisins[0];
	else
		return -1;
}

int deplacement_alea_new(int blocAtLeft, int blocAtRight, int blocAtTop, int blocAtBottom, int blocAtFront, int blocAtBack) {
	return blocAtLeft;
}


/*
int indexFourmiVoisine(int index, , vector<int> &matTransitions) {
	if (!isOnRightBorder(index))
		if (matTransitions[index+1] == index || matTransitions[index+1] == -1*index-2)
			return index+1;
	if (!isOnLeftBorder(index))
		if (matTransitions[index-1] == index || matTransitions[index-1] == -1*index-2)
			return index-1;
	if (!isOnTopBorder(index))
		if (matTransitions[index-taille] == index || matTransitions[index-taille] == -1*index-2)
			return index-taille;
	if (!isOnBottomBorder(index))
		if (matTransitions[index+taille] == index || matTransitions[index+taille] == -1*index-2)
			return index+taille;
	if (!isOnFrontBorder(index)) // A reformuler
		if (matTransitions[index+taille**2] == index || matTransitions[index+taille**2] == -1*index-2)
			return index+taille**2;
	if (!isOnBackBorder(index)) // A reformuler
		if (matTransitions[index-taille**2] == index || matTransitions[index-taille**2] == -1*index-2)
			return index-taille**2;
	return -1;
}*/


//gen�re une matrice al�atoire constitu�e de grains et de bloc accessibles 
//cette matrice n�c�ssite d'�tre retravaill�e par la suite pour la coh�rence des donn�es
struct genereMatrix {
	__host__ __device__
	int operator()(int bloc) {
		
		int states[] = {ACCESSIBLE, GRAIN};
		
		int x = bloc %2;
		return states[x];
	}
};


//retourne le nombre de voisins actifs dans l'entourage de la case � la position "index" dans la matrice
//un voisin actif est une fourmi simple ou une fourmi transportant un bloc
int getNbVoisinsActifs(int index, int left, int right, int top, int bottom, int front, int back) {
	int nb = 0;
	if (!isOnLeftBorder(index)) {
		if (left == FOURMI || left == TRANSIT) 
			nb++;
	}
	if (!isOnRightBorder(index)) {
		if (right == FOURMI || right == TRANSIT) 
			nb++;
	}
	if (!isOnTopBorder(index)) {
		if (top == FOURMI || top == TRANSIT) 
			nb++;
	}
	if (!isOnBottomBorder(index)) {
		if (bottom == FOURMI || bottom == TRANSIT) 
			nb++;
	}
	if (!isOnFrontBorder(index)) {
		if (front == FOURMI || front == TRANSIT) 
			nb++;
	}
	if(!isOnBackBorder(index)) {
		if (back == FOURMI || back == TRANSIT) 
			nb++;
	} 
	return nb;
}

//retourne vrai si la case d'indice "index" est accessible 
bool isAccessible(int index, int left, int right, int top, int bottom, int front, int back) {
	if (!isOnLeftBorder(index)) { 
		return left == GRAIN || left == GRAIN_CONFLIT;
	}
	if (!isOnRightBorder(index)) {
		return right == GRAIN || right == GRAIN_CONFLIT;
	}
	if (!isOnTopBorder(index)) {
		return top == GRAIN || top == GRAIN_CONFLIT;
	}
	if (!isOnBottomBorder(index)) {
		return bottom == GRAIN || bottom == GRAIN_CONFLIT;
	}
	if (!isOnFrontBorder(index)) {
		return front == GRAIN || front == GRAIN_CONFLIT;
	}
	if (!isOnBackBorder(index)) {
		return back == GRAIN || back == GRAIN_CONFLIT;
	}
	return false;
}



/*********************************************
FONCTIONS PRINCIPALES DES BOUCLES DE SIMULATION
*********************************************/

//la fonction updateStates sert � garder l'int�grit� des donn�es pr�sentes dans la matrice
//on v�rifie ici que tous les �tats sont coh�rents vis � vis du mod�le adopt�
//elle prend en param�tres un tuple de 8 valeurs dont :
//- l'indice auquel on se trouve dans la matrice
//- la valeur du bloc courant 
//- les 6 valeurs des des blocs voisins
struct updateStates {

template <typename Tuple>
__host__ __device__
	void operator()(Tuple t) {
		int index = thrust::get<0>(t);
		int bloc = thrust::get<1>(t);
		int blocAtLeft = thrust::get<2>(t);
		int blocAtRight = thrust::get<3>(t);
		int blocAtTop = thrust::get<4>(t);
		int blocAtBottom = thrust::get<5>(t);
		int blocAtFront = thrust::get<6>(t);
		int blocAtBack = thrust::get<7>(t);

		//dans le cas ou le bloc courant est vide ou accessible on v�rifie que ses prorpi�t�s correspondent bien � son �tat
		if(bloc == ACCESSIBLE || bloc == ACCESSIBLE_CONFLIT) {
			if(!isAccessible(index, blocAtLeft, blocAtRight, blocAtTop, blocAtBottom, blocAtFront, blocAtBack))
			thrust::get<8>(t) = VIDE; 
			else {
			if(getNbVoisinsActifs(index, blocAtLeft, blocAtRight, blocAtTop, blocAtBottom, blocAtFront, blocAtBack) > 1)
			thrust::get<8>(t) = ACCESSIBLE_CONFLIT;
			else
			thrust::get<8>(t) = ACCESSIBLE;
			}
		} else if (bloc == GRAIN || bloc == GRAIN_CONFLIT) {
			//Dans le cas ou le bloc est un grain ou du plein il faut v�rifier que ses propri�t�s correspondent bien � son �tat 
			if (getNbVoisinsActifs(index, blocAtLeft, blocAtRight, blocAtTop, blocAtBottom, blocAtFront, blocAtBack) > 1)
			thrust::get<8>(t) = GRAIN_CONFLIT;
			else
			thrust::get<8>(t) = GRAIN;
		} else {
			//si le bloc est de type fourmi ou transit alors il ne change pas d'�tat 
			//on retourne sa valeur inchang�e
			thrust::get<8>(t) = bloc;
		}
	}
	};


/*Cette fonction permet de determiner les intention d'action de toutes les fourmis pr�sentes dans la matrice.
Les actions ne sont pas effectu�es sur la matrice principale, elles sont simplement renseigner.
On cr�� ainsi une matrice suppl�mentaire indiquant les mouvements qui interviendront � l'�tape suivante.
*/
struct transition1 {

	template <typename Tuple>
	__host__ __device__
	void operator() (Tuple t) {
	
		int bloc = thrust::get<0>(t);
		int blocAtLeft = thrust::get<1>(t);
		int blocAtRight = thrust::get<2>(t);
		int blocAtTop = thrust::get<3>(t);
		int blocAtBottom = thrust::get<4>(t);
		int blocAtFront = thrust::get<5>(t);
		int blocAtBack = thrust::get<6>(t);
		
		int choix = rand() % 2;
		if (bloc==FOURMI || bloc==TRANSIT) {
			if (choix==0) { //D�placement
				thrust::get<7>(t) = deplacement_alea_new(blocAtLeft,blocAtRight,blocAtTop,blocAtBottom,blocAtFront,blocAtBack);
			}
			else if (choix==1 && bloc == FOURMI) { //Ramassage
				//vector <int> tmp;
				//tmp.push_back(GRAIN);
				//vector <int> voisins = listeVoisins(index, tmp);
				thrust::get<7>(t) = deplacement_alea_new(blocAtLeft,blocAtRight,blocAtTop,blocAtBottom,blocAtFront,blocAtBack);
			}
			else if (choix==1 && bloc == TRANSIT) { //D�pot
				if (deplacement_alea_new(blocAtLeft,blocAtRight,blocAtTop,blocAtBottom,blocAtFront,blocAtBack)==-1)
					thrust::get<7>(t) = -1;
				else
					thrust::get<7>(t) = -1*deplacement_alea_new(blocAtLeft,blocAtRight,blocAtTop,blocAtBottom,blocAtFront,blocAtBack)-2;
			}
		}
		else
			thrust::get<7>(t) = -1;
	}
};


struct transition2 {

	template <typename Tuple>
	__host__ __device__
	void operator() (Tuple t) {
	
		int blocOriginal = thrust::get<0>(t); // matFourmi[index]
		int blocTransitions = thrust::get<1>(t); //matTransitions[indexFourmi]
		int blocArrivee =  thrust::get<2>(t); // matFourmi[indexFourmi]
		int blocAtLeft = thrust::get<3>(t);
		int blocAtRight = thrust::get<4>(t);
		int blocAtTop = thrust::get<5>(t);
		int blocAtBottom = thrust::get<6>(t);
		int blocAtFront = thrust::get<7>(t);
		int blocAtBack = thrust::get<8>(t);
	
		bool isDeparture = blocOriginal != -1;
		int indexFourmi = indexFourmiVoisine(index);
		bool isArrival = indexFourmi != -1;

		if (isDeparture) {
			if (bloc > -1) //cas d�placement
				thrust::get<9>(t) = ACCESSIBLE;
			else	//cas d�pot
				thrust::get<9>(t) = GRAIN;
		}
		else if (isArrival) {
			if (blocTransitions > -1) { //cas d�placement
				if (blocOriginal == ACCESSIBLE) //cas d�placement simple
					thrust::get<9>(t) = blocArrivee; 
				else if (blocOriginal == GRAIN) //cas ramassage
					thrust::get<9>(t) = TRANSIT;
			}
			else if (blocTransitions < -1)	//cas d�pot
				thrust::get<9>(t) = FOURMI;
		}
		else
			thrust::get<9>(t) = blocOriginal;
	}
};


int main() {
	
	srand ( time(NULL) );
	clock_t t1;
	clock_t t2;
	t1 = clock();
	
	
	// G�n�ration de la matrice
	thrust::host_vector<int> matFourmi(taille*taille*taille);
	thrust::generate(matFourmi.begin(), matFourmi.end(), rand);
	thrust::transform(matFourmi.begin(), matFourmi.end(), matFourmi.begin(), genereMatrix());
	
	// Placement d'une fourmi
	int nbFourmis = 1;
	for (int i = 0 ; i<nbFourmis ; i++) {
		int randvalue = rand() % taille*taille*taille;
		matFourmi[randvalue] = FOURMI;
	}
	

	// Cr�ation des matrices d�cal�es
	int tailleTotale = matFourmi.size();
	thrust::counting_iterator<int> begin(0);
	thrust::counting_iterator<int> end(tailleTotale);

	thrust::host_vector <int> rightIndexes(tailleTotale);
	thrust::host_vector <int> leftIndexes(tailleTotale);
	thrust::host_vector <int> topIndexes(tailleTotale);
	thrust::host_vector <int> bottomIndexes(tailleTotale);
	thrust::host_vector <int> frontIndexes(tailleTotale);
	thrust::host_vector <int> backIndexes(tailleTotale);
	
	//cr�ation des vecteurs contenant les indices d�cal�s du vecteur principal
	thrust::transform(begin, end, leftIndexes.begin(), moveIndex(-1 ,tailleTotale));
	thrust::transform(begin, end, rightIndexes.begin(), moveIndex(1 ,tailleTotale));
	thrust::transform(begin, end, topIndexes.begin(), moveIndex(-taille ,tailleTotale));
	thrust::transform(begin, end, bottomIndexes.begin(), moveIndex(taille ,tailleTotale));
	thrust::transform(begin, end, frontIndexes.begin(), moveIndex(taille*taille ,tailleTotale));
	thrust::transform(begin, end, backIndexes.begin(), moveIndex(-taille*taille ,tailleTotale));
	
	
	//Premi�re mise � jour de la matrice
	//ici, on cr�� des listes d�cal�es pour pouvoir acceder � tous les �l�ments voisins d'un �l�ment particulier
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				begin,
				matFourmi.begin(),
				thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.begin()), 
				thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.begin()), 
				matFourmi.begin()
			)
		),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				end,
				matFourmi.end(), 
				thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.end()), 
				thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.end()), 
				matFourmi.end()
			)
		),
		updateStates()
	);
	
	// Cr�ation de la matrice interm�diaire
	thrust::host_vector <int> matTransitions;
	thrust::fill(matTransitions.begin(), matTransitions.end(), 0);

	cout << "Matrice initiale" << endl;
	printMatrix(matFourmi);
	
	// Demande du nombre d'�tapes � l'utilisateur
	int nbEtapes;
	cout << "Combien d'etapes voulez vous realiser ?" << endl;
	cin >> nbEtapes;
	
	
	// Boucle principale des �tapes
	for (int i=0 ; i<nbEtapes ; i++) {
		cout << "Etape " << i << endl;
		
		cout << "\nMatrice temps" << i << endl;
		
		// Transition 1
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					matFourmi.begin(),
					thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.begin()), 
					matTransitions.begin()
				)
			),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					matFourmi.end(), 
					thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.end()), 
					matTransitions.end()
				)
			),
			transition1()
		);
		cout << "\nMatrice temporaire" << endl;
		printMatrix(matTransitions);
		
		// Transition 2
		/*
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					matTransitions.begin(),
					thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.begin()), 
					thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.begin()), 
					matFourmi.begin()
				)
			),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					matTransitions.end(), 
					thrust::make_permutation_iterator(matFourmi.begin(), leftIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), rightIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), topIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), bottomIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), frontIndexes.end()), 
					thrust::make_permutation_iterator(matFourmi.begin(), backIndexes.end()), 
					matFourmi.end()
				)
			),
			transition1()
		);*/
		
		//matFourmi = updateStatesHost(matFourmi);
		printMatrix(matFourmi);
		
		system("pause");
	}
	
	t2 = clock() - t1;
	cout << "Temps �coul� : " << t2 << endl;

	return 0;
}

/*
nvcc --machine 32 -ccbin "C:\Program Files\Microsoft Visual Studio 10.0\VC\bin"  -I "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v4.0\include" testpara.cu -o testpara
*/
