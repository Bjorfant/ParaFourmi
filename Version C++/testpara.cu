#include "hip/hip_runtime.h"
#include <thrust/for_each.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <iostream>
#include <vector>
#include <cstdlib>

using namespace std;

int taille = 3;


 typedef enum {
	VIDE=0,
	ACCESSIBLE=1,
	ACCESSIBLE_CONFLIT=2,
	GRAIN_CONFLIT=3,
	GRAIN=4,
	FOURMI=5,
	TRANSIT=6
} State;


//retourne vrai si la case d'indice "index" est sur le bord gauche de la matrice

int isOnLeftBorder(int index) {
	return index%taille == 0;
}

//retourne vrai si la case d'indice "index" est sur le bord droit de la matrice 

int isOnRightBorder(int index) {
	return index%taille == taille - 1;
}

//retourne vrai si la case d'indice "index" est sur le bord sup�rieur de la matrice

int isOnTopBorder(int index) {
	return index%(taille*taille) - taille < 0;
}

//retourne vrai si la case d'indice "index" est sur le bord inf�rieur de la matrice

int isOnBottomBorder(int index) {
	return index%(taille*taille) + taille >= taille*taille;
}

//retourne vrai si la case d'indice "index" est sur le bord avant de la matrice 

int isOnFrontBorder(int index) {
	return index + (taille*taille) >= taille*taille*taille;
}

//retourne vrai si la case d'indice "index" est sur le bord arri�re de la matrice

int isOnBackBorder(int index) {
	return index < taille*taille;
}



bool isAccessible(int index, thrust::device_vector<int> matfourmi) {
	// Calcul du nombre de voisins
	if (!isOnRightBorder(index)) {
		if (matfourmi[index+1] == GRAIN)
			return true;
	}
	if (!isOnLeftBorder(index)) {
		if (matfourmi[index-1] == GRAIN)
			return true;
	}
	if (!isOnTopBorder(index)) {
		if (matfourmi[index-taille] == GRAIN)
			return true;
	}
	if (!isOnBottomBorder(index)) {
		if (matfourmi[index+taille] == GRAIN)
			return true;
	}
	if (!isOnFrontBorder(index)) {
		if (matfourmi[index+taille*taille] == GRAIN)
			return true;
	}
	if (!isOnBackBorder(index)) {
		if (matfourmi[index-taille*taille] == GRAIN)
			return true;
	}
	return false;
}


int deplacement_alea(vector <int> voisins) {
	if (voisins.size() >= 1)
		return voisins[0];
	else
		return -1;
}

//r�cup�re les voisins d'une case de la matrice
//possibilit� de filtrer les voisins par une liste d'�tat que l'on cherche 
//si la liste est vide on renvoit tous les voisins

vector <int> listeVoisins(int index, thrust::device_vector <int> filtre, thrust::host_vector<int> &matfourmi) {
	vector <int> voisins;
	bool all = filtre.empty(); //verifie s'il y a une condition
	if (!isOnRightBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index+1]) != filtre.end())
			voisins.push_back(index+1);
	if (!isOnLeftBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index-1]) != filtre.end())
			voisins.push_back(index-1);
	if (!isOnTopBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index-taille]) != filtre.end())
			voisins.push_back(index-taille);
	if (!isOnBottomBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index+taille]) != filtre.end())
			voisins.push_back(index+taille);
	if (!isOnFrontBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index+taille*taille]) != filtre.end())
			voisins.push_back(index+taille*taille);
	if (!isOnBackBorder(index))
		if (all || thrust::find(filtre.begin(), filtre.end(), matfourmi[index-taille*taille]) != filtre.end())
			voisins.push_back(index-taille*taille);
	return voisins;
}


vector <int> listeVoisinsAccessibles(int index, thrust::host_vector<int> &matfourmi) {
	vector <int> v;
	v.push_back(ACCESSIBLE);
	return listeVoisins(index, v, matfourmi);
}


vector <int> listeVoisinsActifs(int index, thrust::host_vector<int> &matfourmi) {
	vector <int> v;
	v.push_back(ACCESSIBLE);
	v.push_back(TRANSIT);
	return listeVoisins(index, v, matfourmi);
}

/*
int indexFourmiVoisine(int index, , vector<int> &matTransitions) {
	if (!isOnRightBorder(index))
		if (matTransitions[index+1] == index || matTransitions[index+1] == -1*index-2)
			return index+1;
	if (!isOnLeftBorder(index))
		if (matTransitions[index-1] == index || matTransitions[index-1] == -1*index-2)
			return index-1;
	if (!isOnTopBorder(index))
		if (matTransitions[index-taille] == index || matTransitions[index-taille] == -1*index-2)
			return index-taille;
	if (!isOnBottomBorder(index))
		if (matTransitions[index+taille] == index || matTransitions[index+taille] == -1*index-2)
			return index+taille;
	if (!isOnFrontBorder(index)) // A reformuler
		if (matTransitions[index+taille**2] == index || matTransitions[index+taille**2] == -1*index-2)
			return index+taille**2;
	if (!isOnBackBorder(index)) // A reformuler
		if (matTransitions[index-taille**2] == index || matTransitions[index-taille**2] == -1*index-2)
			return index-taille**2;
	return -1;
}*/


struct genereMatrix {
	__host__ __device__
	int operator()(int bloc) {
		
		int states[] = {ACCESSIBLE, GRAIN};
		
		int x = bloc %2;
		return states[x];
	}
};

struct placeAnt {
	const int a;

	placeAnt(int _a) : a(_a) {}
	
	__host__ __device__
	int operator()(int bloc) {
		if (bloc == ACCESSIBLE)
			
			//if (a%6 < 2 && getNbFourmi() < 1) {
				//nbFourmi += 1;
				return FOURMI;
			//}
		return bloc;
	}
};



struct updateStates2 {
	
	 template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t) {
		/*t(0) = matfourmi
		t(1) = voisins actifs
		t(2) = isaccessible
		t(3) = matfourmi
		*/
		int bloc = thrust::get<0>(t);
		int nbVoisinsActifs = thrust::get<1>(t);
		bool isAccessible = thrust::get<2>(t);
		
		if (bloc == VIDE || bloc == ACCESSIBLE) {
			if (isAccessible && nbVoisinsActifs<=1)
				thrust::get<3>(t) = ACCESSIBLE;
			else
				thrust::get<3>(t) = VIDE;
		}
		else if (bloc == GRAIN && nbVoisinsActifs>1)
			thrust::get<3>(t) = GRAIN_CONFLIT;
		else if (bloc == GRAIN_CONFLIT && nbVoisinsActifs<=1)
			thrust::get<3>(t) = GRAIN;
		else
			thrust::get<3>(t) = bloc;
		thrust::get<3>(t) = -1;
	}
	
};

thrust::host_vector<int> updateStates (thrust::host_vector<int> &matFourmi) {
	
	// Initialisation de la matrice des voisins actifs ---------- SEQUENTIEL : Modifier la fonction listeVoisinsActifs
	thrust::host_vector<int> matNbVoisinsActifs;
	for(int i=0 ; i<matFourmi.size() ; i++)
		matNbVoisinsActifs.push_back(listeVoisinsActifs(i,matFourmi).size());
		
	// Initialisation de la matrice des bool�ens accessibles ---------- SEQUENTIEL : Modifier la fonction isAccessible
	thrust::host_vector<int> matIsAccessible;
	for(int i=0 ; i<matFourmi.size() ; i++)
		matIsAccessible.push_back(isAccessible(i,matFourmi));
		
	// Application des conditions d'updateStates2 sur les 3 matrices transform�es en tuple (la fonction transform ne prend que 2 elements max)
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(matFourmi.begin(), matNbVoisinsActifs.begin(), matIsAccessible.begin(), matFourmi.begin())
		),
		thrust::make_zip_iterator(
			thrust::make_tuple(matFourmi.end(), matNbVoisinsActifs.end(), matIsAccessible.end(), matFourmi.end())
		),
		updateStates2()
	);
	
	return matFourmi;
}

/*
// index : position dans la matrice
// bloc ; �tat du bloc � la position "index"
int transition(int index, int bloc) {
	int choix = rand() % 2;
	if (bloc==FOURMI || bloc==TRANSIT) {
		if (choix==0) { //D�placement
			voisins = listeVoisinsAccessibles(index);
			return deplacement_alea(voisins);
		}
		else if (choix==1 && bloc == FOURMI) { //Ramassage
			vector <int> tmp;
			tmp.push_back(GRAIN);
			vector <int> voisins = listeVoisins(index, tmp);
			return deplacement_alea(voisins);
		}
		else if (choix==1 && bloc == TRANSIT) { //D�pot
			vector <int> voisins = listeVoisinsAccessibles(index);
			if (deplacement_alea(voisins)==-1)
				return -1;
			else:
				return -1*deplacement_alea(voisins)-2;
		}
	}
	else
		return -1;
}*/
/*
int transition2(int index) {
	int val = matTransitions[index];
	bool isDeparture = val != -1;
	int indexFourmi = indexFourmiVoisine(index);
	bool isArrival = indexFourmi != -1;

	if (isDeparture) {
		if (val > -1) //cas d�placement
			return ACCESSIBLE;
		else	//cas d�pot
			return GRAIN;
	}
	else if (isArrival) {
		if (matTransitions[indexFourmi] > -1) { //cas d�placement
			if (matfourmi[index] == ACCESSIBLE) //cas d�placement simple
				return matfourmi[indexFourmi]; 
			else if (matfourmi[index] == GRAIN) //cas ramassage
				return TRANSIT;
		}
		else if (matTransitions[indexFourmi] < -1) {	//cas d�pot
			return FOURMI;
		else
			cout << "ERREUR DE MERDE" << endl;
	}
	else
		return matfourmi[index];
}
*/

int main() {
	
	int taille = 3;
	
	srand ( time(NULL) );
	
	thrust::host_vector<int> matFourmi(taille*taille*taille);

	clock_t t1;
	clock_t t2;
	t1 = clock();

	t2=clock()-t1;
	t1 = clock();
	
	// G�n�ration de la matrice
	thrust::generate(matFourmi.begin(), matFourmi.end(), rand);
	thrust::transform(matFourmi.begin(), matFourmi.end(), matFourmi.begin(), genereMatrix());
	
	// Placement d'une fourmi -------- A modifier : faire une boucle pour plusieurs fourmis
	int randvalue = rand() % taille*taille*taille;
	matFourmi[randvalue] = FOURMI;
	
	// Mise � jour de la matrice
	matFourmi = updateStates(matFourmi);
	
	t2 = clock() - t1;

	for(int i = 0; i < taille*taille*taille; i++) {
		std::cout<< matFourmi[i] << std::endl;
	}
	return 0;
}

/*
int main() {

	int tailleMatrice = 3;
	int nbEtapes = 0;

	// A reprendre du fichier de tests

	cout << "Matrice initiale" << endl << matfourmi << endl;
	cout << "Combien d'etapes voulez vous realiser ?" << endl;
	cin >> nbEtapes;

	for (int i=0 ; i<nbEtapes ; i++) {
		cout << "Etape " << i << endl;
	}

}*/

/*
nvcc  --machine 32 -ccbin "C:\Program Files\Microsoft Visual Studio 10.0\VC\bin"  -I "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v4.0\include" test.cu -o test
*/